// #include <math.h>

#include <hip/hip_runtime.h>
#include <initializer_list>

// https://developer.nvidia.com/blog/even-easier-introduction-cuda/

// Allows easy switching between float and double.
// #define dtype double
// #define dtype3 double3
#define dtype float
#define dtype3 float3

__device__
const dtype SOFTENING_FACTOR = 0.000000000001f;
__device__
const dtype PI_SQRT_INV = 0.5641895835477563f;
// const double PI_SQRT_INV = 1 / std::sqrt(M_PI);

// todo: Cuda's `threadIdx` can be 1D, 2D, or 3D. 2D may be a better fit here.
// 1D with packing/unpacking is fine, but 2D would be perhaps cleaner. Experiment.

// extern "C" __global__ void matmul(dtype* A, dtype* B, dtype* C, int N) {
//     // Syntax example using 2D inputs.
//     size_t ROW = blockIdx.y * blockDim.y + threadIdx.y;
//     size_t COL = blockIdx.x * blockDim.x + threadIdx.x;
//
//     dtype tmpSum = 0;
//
//     if (ROW < N && COL < N) {
//         // each thread computes one element of the block sub-matrix
//         for (size_t i = 0; i < N; i++) {
//             tmpSum += A[ROW * N + i] * B[i * N + COL];
//         }
//     }
//     C[ROW * N + COL] = tmpSum;
// }


__device__
dtype calc_dist(dtype3 point0, dtype3 point1) {
    dtype3 diff;
    diff.x = point0.x - point1.x;
    diff.y = point0.y - point1.y;
    diff.z = point0.z - point1.z;

    return std::sqrt(diff.x * diff.x + diff.y * diff.y + diff.z * diff.z);
}


__device__
dtype coulomb(dtype3 q0, dtype3 q1, dtype charge) {
    dtype r = calc_dist(q0, q1);

    return 1.f * charge / (r + SOFTENING_FACTOR);
}


// Note that this is for the radial component only, with n=1. Real. See CPU side for a ref.
__device__
dtype sto_val(dtype3 posit_sample, dtype3 posit_nuc, dtype xi, uint8_t n) {
    dtype N = PI_SQRT_INV * std::pow(xi, 1.5);

    dtype r = calc_dist(posit_sample, posit_nuc);

    dtype radial = N * std::pow(r, n - 1) * std::exp(-xi * r / n);
    return radial;
}


// Note that this is for the radial component only, with n=1. Real. See CPU side for a ref.
__device__
dtype sto_second_deriv(dtype3 posit_sample, dtype3 posit_nuc, dtype xi) {
    dtype N = PI_SQRT_INV * pow(xi, 1.5);

    dtype3 diff;
    diff.x = posit_sample.x - posit_nuc.x;
    diff.y = posit_sample.y - posit_nuc.y;
    diff.z = posit_sample.z - posit_nuc.z;

    dtype r = std::sqrt(diff.x * diff.x + diff.y * diff.y + diff.z * diff.z);

    dtype exp_term = std::exp(-xi * r);

    dtype result = 0.;

    for (dtype coord : {diff.x, diff.y, diff.z}) {
        result += std::pow(xi, 2) * std::pow(coord, 2) * exp_term / std::pow(r, 2);
        result += xi * std::pow(coord, 2) * exp_term / std::pow(r, 3);
        result -= xi * exp_term / r;
    }

    dtype radial = N * result;
    return radial;
}



// In this approach, we parallelize operations per sample, but run the
// charge computations in serial, due to the cumulative addition step. This appears
// to be much faster in practice, likely due to the addition being offloaded
// to the CPU in the other approach.
extern "C" __global__
void coulomb_kernel(
    dtype *out,
    dtype3 *posits_charge,
    dtype3 *posits_sample,
    dtype *charges,
    size_t N_charges,
    size_t N_samples
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i_sample = index; i_sample < N_samples; i_sample += stride) {
        // Compute the sum serially, as it may not be possible to naively apply it in parallel,
        // and we may still be saturating GPU cores given the large number of samples.
        for (size_t i_charge = 0; i_charge < N_charges; i_charge++) {
            dtype3 posit_charge = posits_charge[i_charge];
            dtype3 posit_sample = posits_sample[i_sample];

            if (i_sample < N_samples) {
                out[i_sample] += coulomb(posit_charge, posit_sample, charges[i_charge]);
            }
        }
    }
}


extern "C" __global__
void sto_val_kernel(
    dtype *out,
    dtype3 *posits_sample,
    dtype3 posit_nuc,
    dtype xi,
    size_t N_samples
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_samples; i += stride) {
        out[i] = sto_val(posits_sample[i], posit_nuc, xi, 1);
    }
}


// Note that this is for the radial component only, with n=1. Real.
extern "C" __global__
void sto_deriv_kernel(
    dtype *out,
    dtype3 *posits_sample,
    dtype3 posit_nuc,
    dtype xi,
    size_t N_samples
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_samples; i += stride) {
        out[i] = sto_second_deriv(posits_sample[i], posit_nuc, xi);
    }
}

extern "C" __global__
void sto_val_deriv_kernel(
    dtype *out_val,
    dtype *out_second_deriv,
    dtype3 *posits_sample,
    dtype3 posit_nuc,
    dtype xi,
    size_t N_samples
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i_sample = index; i_sample < N_samples; i_sample += stride) {
        out_val[i_sample] += sto_val(posits_sample[i_sample], posit_nuc, xi, 1);
        out_second_deriv[i_sample] += sto_second_deriv(posits_sample[i_sample], posit_nuc, xi);
    }
}

extern "C" __global__
void sto_val_multiple_bases_kernel(
    dtype *out_val,
    dtype3 *posits_sample,
    dtype3 *posits_nuc,
    dtype *xis,
    dtype *weights,
    size_t N_samples,
    size_t N_bases
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i_sample = index; i_sample < N_samples; i_sample += stride) {
        for (size_t i_basis = 0; i_basis < N_bases; i_basis++) {
            out_val[i_sample] += sto_val(posits_sample[i_sample], posits_nuc[i_basis], xis[i_basis], 1) * weights[i_basis];
        }
    }
}


// Combines these 2 operations, as they're likely to be done on the same data set.
extern "C" __global__
void sto_val_deriv_multiple_bases_kernel(
    dtype *out_val,
    dtype *out_second_deriv,
    dtype3 *posits_sample,
    dtype3 posit_nuc,
    dtype *xis,
    dtype *weights,
    size_t N_samples,
    size_t N_bases
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i_sample = index; i_sample < N_samples; i_sample += stride) {
        for (size_t i_basis = 0; i_basis < N_bases; i_basis++) {
            out_val[i_sample] += sto_val(posits_sample[i_sample], posit_nuc, xis[i_basis], 1) * weights[i_basis];
            out_second_deriv[i_sample] += sto_second_deriv(posits_sample[i_sample], posit_nuc, xis[i_basis]) * weights[i_basis];
        }
    }
}
