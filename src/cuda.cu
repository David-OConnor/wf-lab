#include "hip/hip_runtime.h"
// #include <math.h>
#include <initializer_list>

// todo: Header file.
#include "util.cu"


// Note that this is for the radial component only, with n=1. Real. See CPU side for a ref.
__device__
dtype sto_val(dtype3 posit_sample, dtype3 posit_nuc, dtype xi, uint16_t n) {
//     dtype N = PI_SQRT_INV * std::pow(xi, 1.5f);
//
    dtype r = calc_dist(posit_sample, posit_nuc);
//
//     dtype radial = N * std::pow(r, n - 1) * std::exp(-xi * r / n);
//     return radial;

    uint16_t l = 0;

    dtype norm_term_num = std::pow(2. / (n * A_0), 3) * factorial(n - l - 1);
    dtype norm_term_denom = (2 * n * std::pow(factorial(n + l), 3));
    dtype norm_term = std::sqrt(norm_term_num / norm_term_denom);

    dtype exp_term = std::exp(-r / (n * A_0));

    uint16_t lg_l = n - l - 1;
    uint16_t lg_r = 2 * l + 1;
    dtype lg_input = 2.f * r / n;

    dtype polynomial_term = std::pow(2.f * r / n, l) * laguerre(lg_l, lg_r, lg_input);

    return norm_term
        * polynomial_term
        * exp_term;
}


// Note that this is for the radial component only. Real. See CPU side for a ref.
__device__
dtype sto_second_deriv(dtype3 posit_sample, dtype3 posit_nuc, dtype xi, uint16_t n) {
    dtype3 diff;
    diff.x = posit_sample.x - posit_nuc.x;
    diff.y = posit_sample.y - posit_nuc.y;
    diff.z = posit_sample.z - posit_nuc.z;

    dtype r_sq = std::pow(diff.x, 2) + std::pow(diff.y, 2) + std::pow(diff.z, 2);

    if (r_sq < 0.000000001f) {
        return 0.f;
    }

    dtype r = std::sqrt(r_sq);

    uint16_t l = 0;

    dtype exp_term = std::exp(-xi * r / n);
    dtype laguerre_param = 2.f * r / n;

    double norm_term_num = std::pow(2.0f / n, 3) * factorial(n - l - 1);
    double norm_term_denom = std::pow(2 * n * factorial(n + l), 3);
    double norm_term = std::sqrt(norm_term_num / norm_term_denom);

    dtype result = 0.;

    for (auto x : {diff.x, diff.y, diff.z}) {
        double x_sq = std::pow(x, 2);

        if (n == 1 && l == 0) {
            auto term1 = 2.0f * r * ((pow(xi, 2) * x_sq * exp_term) / (pow(n, 2) * r_sq)
                        + (xi * x_sq * exp_term) / (n * pow(r_sq, 1.5f))
                        - (xi * exp_term) / (n * r)) / n;
        
            auto term2 = - (4.0f * xi * x_sq * exp_term) / (pow(n, 2) * r_sq);
        
            auto term3 = (2.0f * (1.0f / r - x_sq / pow(r_sq, 1.5f)) * exp_term) / n;
        
            result += term1 + term2 + term3;
        } else if (n == 2 && l == 0) {
            result += exp_term * 2.f * r / n * (2.f - laguerre_param);
        }

    }

    return result;
}


// In this approach, we parallelize operations per sample, but run the
// charge computations in serial, due to the cumulative addition step. This appears
// to be much faster in practice, likely due to the addition being offloaded
// to the CPU in the other approach.
extern "C" __global__
void coulomb_kernel(
    dtype *out,
    dtype3 *posits_charge,
    dtype3 *posits_sample,
    dtype *charges,
    size_t N_charges,
    size_t N_samples
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i_sample = index; i_sample < N_samples; i_sample += stride) {
        // Compute the sum serially, as it may not be possible to naively apply it in parallel,
        // and we may still be saturating GPU cores given the large number of samples.
        for (size_t i_charge = 0; i_charge < N_charges; i_charge++) {
            dtype3 posit_charge = posits_charge[i_charge];
            dtype3 posit_sample = posits_sample[i_sample];

            if (i_sample < N_samples) {
                out[i_sample] += coulomb(posit_charge, posit_sample, charges[i_charge]);
            }
        }
    }
}


// Note that this is for the radial component only, with n=1. Real.
extern "C" __global__
void sto_val_or_deriv_kernel(
    dtype *out,
    dtype3 *posits_sample,
    dtype3 posit_nuc,
    dtype xi,
    uint16_t n,
//     bool deriv,
    size_t N_samples
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    bool deriv = false; // todo TS OUT OF RESOURCES.

    for (size_t i = index; i < N_samples; i += stride) {
        if (deriv == true) {
            out[i] = sto_second_deriv(posits_sample[i], posit_nuc, xi, n);
        } else {
            out[i] = sto_val(posits_sample[i], posit_nuc, xi, n);
        }
    }
}

// Temp workaround for out-of-resources error on the combined or or and ones.
extern "C" __global__
void sto_deriv_kernel(
    dtype *out,
    dtype3 *posits_sample,
    dtype3 posit_nuc,
    dtype xi,
    uint16_t n,
    size_t N_samples
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i = index; i < N_samples; i += stride) {
        out[i] = sto_second_deriv(posits_sample[i], posit_nuc, xi, n);
    }
}

// We combine value and derivative computations here to reduce IO between host and device.
extern "C" __global__
void sto_val_deriv_kernel(
    dtype *out_val,
    dtype *out_second_deriv,
    dtype3 *posits_sample,
    dtype3 posit_nuc,
    dtype xi,
    uint16_t n,
    size_t N_samples
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i_sample = index; i_sample < N_samples; i_sample += stride) {
        out_val[i_sample] = sto_val(posits_sample[i_sample], posit_nuc, xi, n);
        out_second_deriv[i_sample] = sto_second_deriv(posits_sample[i_sample], posit_nuc, xi, n);
    }
}

extern "C" __global__
void sto_val_multiple_bases_kernel(
    dtype *out_val,
    dtype3 *posits_sample,
    dtype3 *posits_nuc,
    dtype *xis,
    uint16_t *n,
    dtype *weights,
    size_t N_samples,
    size_t N_bases
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i_sample = index; i_sample < N_samples; i_sample += stride) {
        for (size_t i_basis = 0; i_basis < N_bases; i_basis++) {
            out_val[i_sample] += sto_val(posits_sample[i_sample], posits_nuc[i_basis], xis[i_basis], n[i_basis]) * weights[i_basis];
        }
    }
}


// Combines these 2 operations, as they're likely to be done on the same data set.
extern "C" __global__
void sto_val_deriv_multiple_bases_kernel(
    dtype *out_val,
    dtype *out_second_deriv,
    dtype3 *posits_sample,
    dtype3 posit_nuc,
    dtype *xis,
    uint16_t *n,
    dtype *weights,
    size_t N_samples,
    size_t N_bases
) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i_sample = index; i_sample < N_samples; i_sample += stride) {
        for (size_t i_basis = 0; i_basis < N_bases; i_basis++) {
            out_val[i_sample] += sto_val(posits_sample[i_sample], posit_nuc, xis[i_basis], n[i_basis]) * weights[i_basis];
            out_second_deriv[i_sample] += sto_second_deriv(posits_sample[i_sample], posit_nuc, xis[i_basis], n[i_basis]) * weights[i_basis];
        }
    }
}
