
#include <hip/hip_runtime.h>
// This module contains constants and utility functions related to the kernels we use.


// Allows easy switching between float and double.
using dtype = float;
using dtype3 = float3;

__device__
const dtype SOFTENING_FACTOR_SQ = 0.000000000001f;
__device__
const dtype PI_SQRT_INV = 0.5641895835477563f;
__device__
const dtype A_0 = 1.f;
__device__
const dtype EPS_DIV0 = 0.00000000001f;
__device__
const double H = 0.01;
__device__
const double H_SQ = 0.01 * 0.01;

__device__
dtype laguerre(uint16_t n, uint16_t alpha, dtype x) {
    if (n == 0) {
        return 1.f;
    } else if (n == 1) {
        return alpha + 1.f - x;
    } else if (n == 2) {
        return std::pow(x, 2) / 2.f - (alpha + 2.f) * x + (alpha + 1.f) * (alpha + 2.f) / 2.f;
    } else {
        return 0.f; // todo: Implement.
    }
}

__device__
dtype calc_dist(dtype3 point0, dtype3 point1) {
    dtype3 diff;
    diff.x = point0.x - point1.x;
    diff.y = point0.y - point1.y;
    diff.z = point0.z - point1.z;

    return std::sqrt(diff.x * diff.x + diff.y * diff.y + diff.z * diff.z);
}

__device__
uint32_t factorial(uint8_t val) {
    // todo: 64-bit a/r
    if (val == 0) {
        return 1;
    }
    if (val == 1) {
        return 1;
    }
    if (val == 2) {
        return 1;
    }
    if (val == 3) {
        return 6;
    }
    if (val == 4) {
        return 24;
    }
    if (val == 5) {
        return 120;
    }
    if (val == 6) {
        return 720;
    }
    if (val == 7) {
        return 5040;
    }
    if (val == 8) {
        return 40320;
    }
    if (val == 9) {
        return 362880;
    }
    if (val == 10) {
        return 3628800;
    }

    // todo: More A/R
    return 0.;
}


__device__
dtype coulomb(dtype3 q0, dtype3 q1, dtype charge) {
    float r = calc_dist(q0, q1);

    return 1.f * charge / (r + SOFTENING_FACTOR_SQ);
}